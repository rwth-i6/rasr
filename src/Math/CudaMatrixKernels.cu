#include "hip/hip_runtime.h"
/** Copyright 2020 RWTH Aachen University. All rights reserved.
 *
 *  Licensed under the RWTH ASR License (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.hltpr.rwth-aachen.de/rwth-asr/rwth-asr-license.html
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include "CudaMatrixKernels.hh"
#include "stdio.h"

#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __host__
#define __shared__
#endif

#define THREADS_PER_BLOCK 1024

/*
 *
 *  mixed precision axpy
 *
 */

__global__ void __cuda_axpy(int nElements, float alpha, const float* x, double* y) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        y[index] += alpha * x[index];
}

void _cuda_axpy(int nElements, float alpha, const float* x, double* y) {
    int gridSize = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    __cuda_axpy<<<gridSize, THREADS_PER_BLOCK>>>(nElements, alpha, x, y);
}

__global__ void __cuda_axpy(int nElements, double alpha, const double* x, float* y) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        y[index] += alpha * x[index];
}

void _cuda_axpy(int nElements, double alpha, const double* x, float* y) {
    int gridSize = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    __cuda_axpy<<<gridSize, THREADS_PER_BLOCK>>>(nElements, alpha, x, y);
}

__global__ void __cuda_cast(int nElements, const float* x, double* y) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        y[index] = x[index];
}

void _cuda_cast(int nElements, const float* x, double* y) {
    int gridSize = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    __cuda_cast<<<gridSize, THREADS_PER_BLOCK>>>(nElements, x, y);
}

/*
 *
 *  exp
 *
 */
template<typename T>
__global__ void __cuda_exp(T* data, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = exp(data[index]);
}

template<typename T>
void _cuda_exp(T* data, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_exp<<<gridSize, THREADS_PER_BLOCK>>>(data, nElements);
}

template __global__ void __cuda_exp<float>(float*, unsigned int);
template __global__ void __cuda_exp<double>(double*, unsigned int);
template void            _cuda_exp<float>(float*, unsigned int, unsigned int);
template void            _cuda_exp<double>(double*, unsigned int, unsigned int);

/*
 *
 *  log
 *
 */

template<typename T>
__global__ void __cuda_log(T* data, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = log(data[index]);
}

template<typename T>
void _cuda_log(T* data, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_log<<<gridSize, THREADS_PER_BLOCK>>>(data, nElements);
}

template __global__ void __cuda_log<float>(float*, unsigned int);
template __global__ void __cuda_log<double>(double*, unsigned int);
template void            _cuda_log<float>(float*, unsigned int, unsigned int);
template void            _cuda_log<double>(double*, unsigned int, unsigned int);

/*
 *
 *  pow
 *
 */

template<typename T>
__global__ void __cuda_pow(T* data, unsigned int nElements, T exponent) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = pow(data[index], exponent);
}

template<typename T>
void _cuda_pow(T* data, unsigned int nRows, unsigned int nColumns, T exponent) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    __cuda_pow<<<gridSize, THREADS_PER_BLOCK>>>(data, nElements, exponent);
}

template void _cuda_pow<float>(float*, unsigned int, unsigned int, float);
template void _cuda_pow<double>(double*, unsigned int, unsigned int, double);

/*
 *
 * tanh
 *
 *
 */

template<typename T>
__global__ void __cuda_tanh(T* data, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = tanh(data[index]);
}

template<typename T>
void _cuda_tanh(T* data, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_tanh<<<gridSize, THREADS_PER_BLOCK>>>(data, nElements);
}

template __global__ void __cuda_tanh<float>(float*, unsigned int);
template __global__ void __cuda_tanh<double>(double*, unsigned int);
template void            _cuda_tanh<float>(float*, unsigned int, unsigned int);
template void            _cuda_tanh<double>(double*, unsigned int, unsigned int);

/*
 *
 * sigmoid
 *
 */

template<typename T>
__global__ void __cuda_sigmoid1(T* data, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = 1.0 / (1.0 + exp(-data[index]));
}

template<typename T>
__global__ void __cuda_sigmoid(T gamma, T* data, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = 1.0 / (1.0 + exp(-gamma * data[index]));
}

template<typename T>
void _cuda_sigmoid(T gamma, T* data, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    if (gamma == 1.0)
        __cuda_sigmoid1<<<gridSize, THREADS_PER_BLOCK>>>(data, nElements);
    else
        __cuda_sigmoid<<<gridSize, THREADS_PER_BLOCK>>>(gamma, data, nElements);
}

template void            _cuda_sigmoid<double>(double gamma, double* data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_sigmoid<double>(double gamma, double* data, unsigned int nElements);
template __global__ void __cuda_sigmoid1<double>(double* data, unsigned int nElements);
template void            _cuda_sigmoid<float>(float gamma, float* data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_sigmoid<float>(float gamma, float* data, unsigned int nElements);
template __global__ void __cuda_sigmoid1<float>(float* data, unsigned int nElements);

/*
 *
 * softmax
 *
 */

template<typename T>
__global__ void __cuda_softmax(T* data, unsigned int nRows, unsigned int nColumns) {
    unsigned int column        = blockIdx.x;
    unsigned int tid           = threadIdx.x;
    unsigned int blocksize     = (unsigned int)ceil((float)nRows / blockDim.x);  // e.g. 4501/1024 = 5
    unsigned int max_thread_id = (unsigned int)floor((float)nRows / blocksize);  // e.g. 4501/5    = 900

    volatile __shared__ T tmp[THREADS_PER_BLOCK];
    volatile __shared__ T max;
    tmp[tid] = -9999999999;
    T val;

    //// step 1: find maximum in the column
    // each thread finds a maximum in its own "block"
    if (column < nColumns && tid <= max_thread_id) {
        uint beginCol = column * nRows;
        for (uint i = tid * blocksize; i < (tid + 1) * blocksize; ++i) {
            if (i >= nRows)
                break;
            val = data[beginCol + i];
            if (val > tmp[tid])
                tmp[tid] = val;
        }
    }
    __syncthreads();

    // max-reduction
    for (uint s = THREADS_PER_BLOCK / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (tmp[tid + s] > tmp[tid])
                tmp[tid] = tmp[tid + s];
        }
        __syncthreads();
    }

    // one thread stores the maximum in the shared memory
    if (tid == 0 && column < nColumns) {
        max = tmp[0];
    }
    __syncthreads();
    tmp[tid] = 0;

    //// step 2: subtract max from each value and store the sum of
    ////         exp(x-max) in the shared memory
    if (column < nColumns && tid <= max_thread_id) {
        uint beginCol = column * nRows;
        for (uint i = tid * blocksize; i < (tid + 1) * blocksize; ++i) {
            if (i >= nRows)
                break;
            val                = exp(data[beginCol + i] - max);
            data[beginCol + i] = val;
            tmp[tid] += val;
        }
    }
    __syncthreads();

    // sum-reduction; the result is in tmp[0] (softmax normalization)
    for (uint s = THREADS_PER_BLOCK / 2; s > 0; s >>= 1) {
        if (tid < s) {
            tmp[tid] += tmp[tid + s];
        }
        __syncthreads();
    }

    //// step 3: normalize each entry in the column
    if (column < nColumns && tid <= max_thread_id) {
        uint beginCol = column * nRows;
        for (uint i = tid * blocksize; i < (tid + 1) * blocksize; ++i) {
            if (i >= nRows)
                break;
            data[beginCol + i] /= tmp[0];
        }
    }
}

template<typename T>
void _cuda_softmax(T* data, unsigned int nRows, unsigned int nColumns) {
    __cuda_softmax<<<nColumns, THREADS_PER_BLOCK>>>(data, nRows, nColumns);
}

template __global__ void __cuda_softmax(double* data, unsigned int nRows, unsigned int nColumns);
template void            _cuda_softmax(double* data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_softmax(float* data, unsigned int nRows, unsigned int nColumns);
template void            _cuda_softmax(float* data, unsigned int nRows, unsigned int nColumns);

/*
 *
 * addSummedRows
 *
 */
template<typename T>
__global__ void __cuda_addSummedRows(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale) {
    unsigned int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (columnIndex < nColumns) {
        float result = 0.0;
        for (unsigned int i = 0; i < nRows; i++) {
            // result += matrix(i,columnIndex)
            result += matrixDevPtr[columnIndex * nRows + i];
        }
        vectorDevPtr[columnIndex] += scale * result;
    }
}

template<typename T>
void _cuda_addSummedRows(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale) {
    // parallelize over columns
    int gridSize = (int)ceil((float)nColumns / THREADS_PER_BLOCK);

    __cuda_addSummedRows<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nColumns, scale);
}

template __global__ void __cuda_addSummedRows(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template void            _cuda_addSummedRows(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template __global__ void __cuda_addSummedRows(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template void            _cuda_addSummedRows(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);

/*
 * slightly faster version using tmp array
 *
 */
template<typename T>
__global__ void __cuda_summedRowsTmp(const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                     T* tmpDevPtr, unsigned int tmpRows) {
    unsigned int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int columnPart  = blockIdx.y;
    if (columnIndex < nColumns) {
        unsigned int nRowsDiv = nRows / tmpRows;
        unsigned int startRow = columnPart * nRowsDiv;
        if (startRow < nRows) {
            unsigned int endRow = columnPart == tmpRows - 1 ? nRows : (columnPart + 1) * nRowsDiv;
            T            result = 0.0;
            for (unsigned int i = startRow; i < endRow; i++) {
                // result += matrix(i, columnIndex)
                result += matrixDevPtr[columnIndex * nRows + i];
            }
            tmpDevPtr[columnIndex * tmpRows + columnPart] = result;
        }
    }
}

template<typename T>
void _cuda_addSummedRows(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                         T* tmpDevPtr, unsigned int tmpRows, const T scale) {
    int  gridDimx = (int)ceil((float)nColumns / THREADS_PER_BLOCK);
    int  gridDimy = tmpRows;
    dim3 gridSize(gridDimx, gridDimy);
    __cuda_summedRowsTmp<<<gridSize, THREADS_PER_BLOCK>>>(matrixDevPtr, nRows, nColumns, tmpDevPtr, tmpRows);

    _cuda_addSummedRows<T>(vectorDevPtr, tmpDevPtr, tmpRows, nColumns, scale);
}

template __global__ void __cuda_summedRowsTmp<double>(const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                                      double* tmpDevPtr, unsigned int tmpRows);
template void            _cuda_addSummedRows<double>(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                                     double* tmpDevPtr, unsigned int tmpRows, const double scale);
template __global__ void __cuda_summedRowsTmp<float>(const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                                     float* tmpDevPtr, unsigned int tmpRows);
template void            _cuda_addSummedRows<float>(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                                    float* tmpDevPtr, unsigned int tmpRows, const float scale);
/*
 *
 * addSummedColumns
 *
 */

template<typename T, typename S>
__global__ void __cuda_addSummedColumns(T* vectorDevPtr, const S* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const S scale) {
    unsigned int rowIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (rowIndex < nRows) {
        T result = 0.0;
        for (unsigned int i = 0; i < nColumns; i++) {
            // result += matrix(rowIndex,i)
            result += matrixDevPtr[i * nRows + rowIndex];
        }
        vectorDevPtr[rowIndex] += scale * result;
    }
}

template<typename T, typename S>
void _cuda_addSummedColumns(T* vectorDevPtr, const S* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const S scale) {
    // parallelize over rows
    int gridSize = (int)ceil((float)nRows / THREADS_PER_BLOCK);

    __cuda_addSummedColumns<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nColumns, scale);
}

template __global__ void __cuda_addSummedColumns<double, double>(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template void            _cuda_addSummedColumns<double, double>(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template __global__ void __cuda_addSummedColumns<float, float>(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template void            _cuda_addSummedColumns<float, float>(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template __global__ void __cuda_addSummedColumns<double, float>(double* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template void            _cuda_addSummedColumns<double, float>(double* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);

/*
 *
 * addSquaredSummedColumns
 *
 */

template<typename T>
__global__ void __cuda_addSquaredSummedColumns(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale) {
    unsigned int rowIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (rowIndex < nRows) {
        T result = 0.0;
        for (unsigned int i = 0; i < nColumns; i++) {
            result += matrixDevPtr[i * nRows + rowIndex] * matrixDevPtr[i * nRows + rowIndex];
        }
        vectorDevPtr[rowIndex] += scale * result;
    }
}

template<typename T>
void _cuda_addSquaredSummedColumns(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const T scale) {
    // parallelize over rows
    int gridSize = (int)ceil((float)nRows / THREADS_PER_BLOCK);

    __cuda_addSquaredSummedColumns<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nColumns, scale);
}

template __global__ void __cuda_addSquaredSummedColumns(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template void            _cuda_addSquaredSummedColumns(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const double scale);
template __global__ void __cuda_addSquaredSummedColumns(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);
template void            _cuda_addSquaredSummedColumns(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns, const float scale);

/*
 *
 * elementwise multiplication
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplication(T* data, T* datab, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = data[index] * datab[index];
}

template<typename T>
void _cuda_elementwiseMultiplication(T* data, T* datab, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_elementwiseMultiplication<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements);
}

template __global__ void __cuda_elementwiseMultiplication<double>(double* data, double* datab, unsigned int nElements);
template __global__ void __cuda_elementwiseMultiplication<float>(float* data, float* datab, unsigned int nElements);
template void            _cuda_elementwiseMultiplication<double>(double* data, double* datab, unsigned int nRows, unsigned int nColumns);
template void            _cuda_elementwiseMultiplication<float>(float* data, float* datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * elementwise division
 *
 */

template<typename T>
__global__ void __cuda_elementwiseDivision(T* data, T* datab, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = data[index] / datab[index];
}

template<typename T>
void _cuda_elementwiseDivision(T* data, T* datab, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_elementwiseDivision<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements);
}

template __global__ void __cuda_elementwiseDivision<double>(double* data, double* datab, unsigned int nElements);
template __global__ void __cuda_elementwiseDivision<float>(float* data, float* datab, unsigned int nElements);
template void            _cuda_elementwiseDivision<double>(double* data, double* datab, unsigned int nRows, unsigned int nColumns);
template void            _cuda_elementwiseDivision<float>(float* data, float* datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * add constant elementwise
 *
 */
template<typename T>
__global__ void __cuda_addConstantElementwise(T constant, T* data, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = data[index] + constant;
}

template<typename T>
void _cuda_addConstantElementwise(T constant, T* data, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((T)nElements / THREADS_PER_BLOCK);

    __cuda_addConstantElementwise<<<gridSize, THREADS_PER_BLOCK>>>(constant, data, nElements);
}

template __global__ void __cuda_addConstantElementwise<double>(double constant, double* data, unsigned int nElements);
template void            _cuda_addConstantElementwise<double>(double constant, double* data, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_addConstantElementwise<float>(float constant, float* data, unsigned int nElements);
template void            _cuda_addConstantElementwise<float>(float constant, float* data, unsigned int nRows, unsigned int nColumns);

/*
 *
 * getMaxOfColumns
 *
 */
template<typename T>
__global__ void __cuda_getMaxOfColumns(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns) {
    unsigned int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (columnIndex < nColumns) {
        T result = 0.0;
        for (unsigned int i = 0; i < nRows; i++) {
            // result += matrix(i, columnIndex)
            T val  = matrixDevPtr[columnIndex * nRows + i];
            result = fmax(result, val);
        }
        vectorDevPtr[columnIndex] = result;
    }
}

template<typename T>
void _cuda_getMaxOfColumns(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns) {
    // parallelize over columns
    int gridSize = (int)ceil((float)nColumns / THREADS_PER_BLOCK);

    __cuda_getMaxOfColumns<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nColumns);
}

template __global__ void __cuda_getMaxOfColumns(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template void            _cuda_getMaxOfColumns(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_getMaxOfColumns(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template void            _cuda_getMaxOfColumns(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 * slightly faster version using tmp array
 */

template<typename T>
__global__ void __cuda_getMaxOfColumnsTmp(const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                          T* tmpDevPtr, unsigned int tmpRows) {
    unsigned int columnIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int columnPart  = blockIdx.y;
    if (columnIndex < nColumns) {
        unsigned int nRowsDiv = nRows / tmpRows;
        unsigned int startRow = columnPart * nRowsDiv;
        if (startRow < nRows) {
            unsigned int endRow = columnPart == tmpRows - 1 ? nRows : (columnPart + 1) * nRowsDiv;
            T            result = 0.0;
            for (unsigned int i = startRow; i < endRow; i++) {
                // result += matrix(i, columnIndex)
                T val  = matrixDevPtr[columnIndex * nRows + i];
                result = fmax(result, val);
            }
            tmpDevPtr[columnIndex * tmpRows + columnPart] = result;
        }
    }
}

template<typename T>
void _cuda_getMaxOfColumns(T* vectorDevPtr, const T* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                           T* tmpDevPtr, unsigned int tmpRows) {
    int  gridDimx = (int)ceil((float)nColumns / THREADS_PER_BLOCK);
    int  gridDimy = tmpRows;
    dim3 gridSize(gridDimx, gridDimy);

    __cuda_getMaxOfColumnsTmp<<<gridSize, THREADS_PER_BLOCK>>>(matrixDevPtr, nRows, nColumns, tmpDevPtr, tmpRows);

    _cuda_getMaxOfColumns<T>(vectorDevPtr, tmpDevPtr, tmpRows, nColumns);
}

template __global__ void __cuda_getMaxOfColumnsTmp(const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                                   double* tmpDevPtr, unsigned int tmpRows);
template void            _cuda_getMaxOfColumns(double* vectorDevPtr, const double* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                               double* tmpDevPtr, unsigned int tmpRows);
template __global__ void __cuda_getMaxOfColumnsTmp(const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                                   float* tmpDevPtr, unsigned int tmpRows);
template void            _cuda_getMaxOfColumns(float* vectorDevPtr, const float* matrixDevPtr, unsigned int nRows, unsigned int nColumns,
                                               float* tmpDevPtr, unsigned int tmpRows);
/*
 *
 * elementwiseMultiplicationWithSigmoidDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithSigmoidDerivative(T* data, T* datab, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = data[index] * (datab[index] * (1 - datab[index]));
}

template<typename T>
void _cuda_elementwiseMultiplicationWithSigmoidDerivative(T* data, T* datab, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_elementwiseMultiplicationWithSigmoidDerivative<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements);
}

template __global__ void __cuda_elementwiseMultiplicationWithSigmoidDerivative(double* data, double* datab, unsigned int nElements);
template void            _cuda_elementwiseMultiplicationWithSigmoidDerivative(double* data, double* datab, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_elementwiseMultiplicationWithSigmoidDerivative(float* data, float* datab, unsigned int nElements);
template void            _cuda_elementwiseMultiplicationWithSigmoidDerivative(float* data, float* datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * elementwiseMultiplicationWithTanhDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithTanhDerivative(T* data, T* datab, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = data[index] * (1 - pow(datab[index], 2));
}

template<typename T>
void _cuda_elementwiseMultiplicationWithTanhDerivative(T* data, T* datab, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_elementwiseMultiplicationWithTanhDerivative<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements);
}

template __global__ void __cuda_elementwiseMultiplicationWithTanhDerivative(double* data, double* datab, unsigned int nElements);
template void            _cuda_elementwiseMultiplicationWithTanhDerivative(double* data, double* datab, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_elementwiseMultiplicationWithTanhDerivative(float* data, float* datab, unsigned int nElements);
template void            _cuda_elementwiseMultiplicationWithTanhDerivative(float* data, float* datab, unsigned int nRows, unsigned int nColumns);

/*
 *
 * multiplicationWithSoftmaxDerivative
 *
 */

template<typename T>
__global__ void __cuda_multiplicationWithSoftmaxDerivative(T* data, T* datab, T* datac, unsigned int nElements, unsigned int nRows) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = datab[index] * (data[index] - datac[index / nRows]);
}

template<typename T>
void _cuda_multiplicationWithSoftmaxDerivative(T* data, T* datab, T* datac, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_multiplicationWithSoftmaxDerivative<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, datac, nElements, nRows);
}

template __global__ void __cuda_multiplicationWithSoftmaxDerivative(double* data, double* datab, double* datac, unsigned int nElements, unsigned int nRows);
template void            _cuda_multiplicationWithSoftmaxDerivative(double* data, double* datab, double* datac, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_multiplicationWithSoftmaxDerivative(float* data, float* datab, float* datac, unsigned int nElements, unsigned int nRows);
template void            _cuda_multiplicationWithSoftmaxDerivative(float* data, float* datab, float* datac, unsigned int nRows, unsigned int nColumns);

/*
 * elementwiseMultiplicationWithRectifiedDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithRectifiedDerivative(T* errOut, T* activations, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        if (activations[index] <= 0)
            errOut[index] = 0;
}
template<typename T>
void _cuda_elementwiseMultiplicationWithRectifiedDerivative(T* data, T* datab, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    __cuda_elementwiseMultiplicationWithRectifiedDerivative<T><<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements);
}
template __global__ void __cuda_elementwiseMultiplicationWithRectifiedDerivative<float>(float*, float*, unsigned int);
template __global__ void __cuda_elementwiseMultiplicationWithRectifiedDerivative<double>(double*, double*, unsigned int);
template void            _cuda_elementwiseMultiplicationWithRectifiedDerivative<float>(float*, float*, unsigned int, unsigned int);
template void            _cuda_elementwiseMultiplicationWithRectifiedDerivative<double>(double*, double*, unsigned int, unsigned int);

/*
 * elementwiseMultiplicationWithEluDerivative
 *
 */

template<typename T>
__global__ void __cuda_elementwiseMultiplicationWithEluDerivative(T* errOut, T* activations, T alpha, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements && activations[index] < 0) {
        errOut[index] *= (activations[index] + alpha);
    }
}
template<typename T>
void _cuda_elementwiseMultiplicationWithEluDerivative(T* data, T* datab, T alpha, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    __cuda_elementwiseMultiplicationWithEluDerivative<T><<<gridSize, THREADS_PER_BLOCK>>>(data, datab, alpha, nElements);
}
template __global__ void __cuda_elementwiseMultiplicationWithEluDerivative<float>(float*, float*, float, unsigned int);
template __global__ void __cuda_elementwiseMultiplicationWithEluDerivative<double>(double*, double*, double, unsigned int);
template void            _cuda_elementwiseMultiplicationWithEluDerivative<float>(float*, float*, float, unsigned int, unsigned int);
template void            _cuda_elementwiseMultiplicationWithEluDerivative<double>(double*, double*, double, unsigned int, unsigned int);

/*
 *
 * addToAllColumns
 *
 */

template<typename T>
__global__ void __cuda_addToAllColumns(T* data, T* datab, unsigned int nElements, unsigned int nRows, T alpha) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] += alpha * datab[index % nRows];
}

template<typename T>
void _cuda_addToAllColumns(T* data, T* datab, unsigned int nRows, unsigned int nColumns, T alpha) {
    // TODO implement kernel without % operator (slow on GPU)
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_addToAllColumns<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements, nRows, alpha);
}

template __global__ void __cuda_addToAllColumns<double>(double* data, double* datab, unsigned int nElements, unsigned int nRows, double alpha);
template void            _cuda_addToAllColumns<double>(double* data, double* datab, unsigned int nRows, unsigned int nColumns, double alpha);
template __global__ void __cuda_addToAllColumns<float>(float* data, float* datab, unsigned int nElements, unsigned int nRows, float alpha);
template void            _cuda_addToAllColumns<float>(float* data, float* datab, unsigned int nRows, unsigned int nColumns, float alpha);

// with offset (caller has to add offset to data and adjust nRowsMat)

template<typename T>
__global__ void __cuda_addToAllColumnsWithOffset(T* data, T* datab, unsigned int nElements, unsigned int nRowsMat, unsigned int nRowsVec, T alpha) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int col   = index / nRowsVec;
    unsigned int row   = index % nRowsVec;
    if (index < nElements) {
        data[col * nRowsMat + row] += alpha * datab[row];
    }
}

template<typename T>
void _cuda_addToAllColumnsWithOffset(T* data, T* datab, unsigned int nRowsMat, unsigned nRowsVec, unsigned int nColumns, T alpha) {
    // TODO implement kernel without % operator (slow on GPU)
    unsigned int nElements = nRowsVec * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_addToAllColumns<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements, nRowsMat, nRowsVec, alpha);
}

template __global__ void __cuda_addToAllColumnsWithOffset<double>(double* data, double* datab, unsigned int nElements, unsigned int nRowsMat, unsigned int nRowsVec, double alpha);
template void            _cuda_addToAllColumnsWithOffset<double>(double* data, double* datab, unsigned int nRowsMat, unsigned int nRowsVec, unsigned int nColumns, double alpha);
template __global__ void __cuda_addToAllColumnsWithOffset<float>(float* data, float* datab, unsigned int nElements, unsigned int nRowsMat, unsigned int nRowsVec, float alpha);
template void            _cuda_addToAllColumnsWithOffset<float>(float* data, float* datab, unsigned int nRowsMat, unsigned int nRowsVec, unsigned int nColumns, float alpha);

/*
 *
 * addToAllRows
 *
 */
template<typename T>
__global__ void __cuda_addToAllRows(T* data, T* datab, unsigned int nElements, unsigned int nRows, T alpha) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] += alpha * datab[index / nRows];
}
template<typename T>
void _cuda_addToAllRows(T* data, T* datab, unsigned int nRows, unsigned int nColumns, T alpha) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_addToAllRows<<<gridSize, THREADS_PER_BLOCK>>>(data, datab, nElements, nRows, alpha);
}

template __global__ void __cuda_addToAllRows<double>(double* data, double* datab, unsigned int nElements, unsigned int nRows, double alpha);
template void            _cuda_addToAllRows<double>(double* data, double* datab, unsigned int nRows, unsigned int nColumns, double alpha);
template __global__ void __cuda_addToAllRows<float>(float* data, float* datab, unsigned int nElements, unsigned int nRows, float alpha);
template void            _cuda_addToAllRows<float>(float* data, float* datab, unsigned int nRows, unsigned int nColumns, float alpha);

/*
 *
 * multiplyColumnsByScalars
 *
 */
template<typename T>
__global__ void __cuda_multiplyColumnsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nElements) {
    unsigned int index    = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int colIndex = index / nRows;
    if (index < nElements)
        matrixDevPtr[index] = matrixDevPtr[index] * vectorDevPtr[colIndex];
}
template<typename T>
void _cuda_multiplyColumnsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_multiplyColumnsByScalars<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nElements);
}

template __global__ void __cuda_multiplyColumnsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_multiplyColumnsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_multiplyColumnsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_multiplyColumnsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 * divideColumnsByScalars
 *
 */
template<typename T>
__global__ void __cuda_divideColumnsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nElements) {
    unsigned int index    = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int colIndex = index / nRows;
    if (index < nElements)
        matrixDevPtr[index] = matrixDevPtr[index] / vectorDevPtr[colIndex];
}
template<typename T>
void _cuda_divideColumnsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_divideColumnsByScalars<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nElements);
}

template __global__ void __cuda_divideColumnsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_divideColumnsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_divideColumnsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_divideColumnsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 * multiplyRowsByScalars
 *
 */
template<typename T>
__global__ void __cuda_multiplyRowsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nElements) {
    unsigned int index    = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int rowIndex = index % nRows;
    if (index < nElements)
        matrixDevPtr[index] = matrixDevPtr[index] * vectorDevPtr[rowIndex];
}
template<typename T>
void _cuda_multiplyRowsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_multiplyRowsByScalars<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nElements);
}

template __global__ void __cuda_multiplyRowsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_multiplyRowsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_multiplyRowsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_multiplyRowsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 * divideRowsByScalars
 *
 */
template<typename T>
__global__ void __cuda_divideRowsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nElements) {
    unsigned int index    = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int rowIndex = index % nRows;
    if (index < nElements)
        matrixDevPtr[index] = matrixDevPtr[index] / vectorDevPtr[rowIndex];
}
template<typename T>
void _cuda_divideRowsByScalars(const T* vectorDevPtr, T* matrixDevPtr, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_divideRowsByScalars<<<gridSize, THREADS_PER_BLOCK>>>(vectorDevPtr, matrixDevPtr, nRows, nElements);
}

template __global__ void __cuda_divideRowsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_divideRowsByScalars<double>(const double* vectorDevPtr, double* matrixDevPtr, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_divideRowsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nElements);
template void            _cuda_divideRowsByScalars<float>(const float* vectorDevPtr, float* matrixDevPtr, unsigned int nRows, unsigned int nColumns);

/*
 *
 * sign
 *
 */
template<typename T>
__global__ void __cuda_sign(T* out, const T* in, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        out[index] = in[index] == 0 ? 0 : copysignf(1.0, in[index]);
    }
}
template<typename T>
void _cuda_sign(T* out, const T* in, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_sign<<<gridSize, THREADS_PER_BLOCK>>>(out, in, nElements);
}

template __global__ void __cuda_sign<double>(double* out, const double* in, unsigned int nElements);
template void            _cuda_sign<double>(double* out, const double* in, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_sign<float>(float* out, const float* in, unsigned int nElements);
template void            _cuda_sign<float>(float* out, const float* in, unsigned int nRows, unsigned int nColumns);

/*
 *
 *  fill
 *
 */
template<typename T>
__global__ void __cuda_fill(T* data, T value, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements)
        data[index] = value;
}
template<typename T>
void _cuda_fill(T* data, T value, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_fill<<<gridSize, THREADS_PER_BLOCK>>>(data, value, nElements);
}

template __global__ void __cuda_fill<double>(double* data, double value, unsigned int nElements);
template void            _cuda_fill<double>(double* data, double value, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_fill<float>(float* data, float value, unsigned int nElements);
template void            _cuda_fill<float>(float* data, float value, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_fill<uint>(uint* data, uint value, unsigned int nElements);
template void            _cuda_fill<uint>(uint* data, uint value, unsigned int nRows, unsigned int nColumns);

/*
 *
 *  ensure minimal value
 *
 */
template<typename T>
__global__ void __cuda_ensureMinimalValue(T* data, T value, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ((index < nElements) && (data[index] < value))
        data[index] = value;
}

template<typename T>
void _cuda_ensureMinimalValue(T* data, T value, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_ensureMinimalValue<<<gridSize, THREADS_PER_BLOCK>>>(data, value, nElements);
}

template __global__ void __cuda_ensureMinimalValue(double* data, double value, unsigned int nElements);
template void            _cuda_ensureMinimalValue(double* data, double value, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_ensureMinimalValue(float* data, float value, unsigned int nElements);
template void            _cuda_ensureMinimalValue(float* data, float value, unsigned int nRows, unsigned int nColumns);

/*
 *
 *  ELU
 *
 */
template<typename T>
__global__ void __cuda_elu(T* data, T alpha, unsigned int nElements) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements && data[index] < 0)
        data[index] = alpha * (exp(data[index]) - 1);
}

template<typename T>
void _cuda_elu(T* data, T alpha, unsigned int nRows, unsigned int nColumns) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);
    __cuda_elu<<<gridSize, THREADS_PER_BLOCK>>>(data, alpha, nElements);
}

template __global__ void __cuda_elu(double* data, double alpha, unsigned int nElements);
template void            _cuda_elu(double* data, double alpha, unsigned int nRows, unsigned int nColumns);
template __global__ void __cuda_elu(float* data, float alpha, unsigned int nElements);
template void            _cuda_elu(float* data, float alpha, unsigned int nRows, unsigned int nColumns);

/*
 *
 * nClassificationErrors
 *
 *
 */
template<typename T>
__global__ void __cuda_nClassificationErrors(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* errBuf) {
    unsigned int column        = blockIdx.x;
    unsigned int tid           = threadIdx.x;
    unsigned int blocksize     = (unsigned int)ceil((float)nRows / blockDim.x);  // e.g. 4501/1024 = 5
    unsigned int max_thread_id = (unsigned int)floor((float)nRows / blocksize);  // e.g. 4501/5    = 900

    volatile __shared__ bool error_found[THREADS_PER_BLOCK];
    error_found[tid] = false;

    if (column < nColumns && tid <= max_thread_id) {
        uint beginCol  = column * nRows;
        uint c_true    = alignmentDevPtr[column];
        T    true_prob = matrixPtr[beginCol + c_true];

        for (uint i = tid * blocksize; i < (tid + 1) * blocksize; ++i) {
            if (i >= nRows)
                break;
            if (i == c_true)
                continue;
            error_found[tid] |= matrixPtr[beginCol + i] > true_prob;
        }
    }
    __syncthreads();

    for (uint s = THREADS_PER_BLOCK / 2; s > 0; s >>= 1) {
        if (tid < s)
            error_found[tid] |= error_found[tid + s];
        __syncthreads();
    }
    if (tid == 0 && column < nColumns) {
        errBuf[column] = error_found[0] ? 1.0 : 0.0;
    }
}
template<typename T>
void _cuda_nClassificationErrors(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* errorBuf) {
    __cuda_nClassificationErrors<<<nColumns, THREADS_PER_BLOCK>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, errorBuf);
}

template __global__ void __cuda_nClassificationErrors<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* errBuf);
template void            _cuda_nClassificationErrors<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* errBuf);
template __global__ void __cuda_nClassificationErrors<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* errBuf);
template void            _cuda_nClassificationErrors<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* errBuf);

// crossEntropyObjectiveFunction
template<typename T>
__global__ void __cuda_crossEntropyObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn) {
    *objFctn = 0.0f;
    for (int column = 0; column < nColumns; column++) {
        unsigned int position = column * nRows + alignmentDevPtr[column];
        *objFctn -= log(matrixPtr[position]);
    }
}

template<typename T>
void _cuda_crossEntropyObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn) {
    // no parallelization, but probably not relevant
    __cuda_crossEntropyObjectiveFunction<<<1, 1>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, objFctn);
}

template __global__ void __cuda_crossEntropyObjectiveFunction<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn);
template void            _cuda_crossEntropyObjectiveFunction<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn);
template __global__ void __cuda_crossEntropyObjectiveFunction<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn);
template void            _cuda_crossEntropyObjectiveFunction<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn);

// weightedCrossEntropyObjectiveFunction
template<typename T>
__global__ void __cuda_weightedCrossEntropyObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn, T* weights) {
    *objFctn = 0.0f;
    for (int column = 0; column < nColumns; column++) {
        unsigned int position = column * nRows + alignmentDevPtr[column];
        *objFctn -= log(matrixPtr[position]) * weights[column];
    }
}

template<typename T>
void _cuda_weightedCrossEntropyObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn, T* weights) {
    // no parallelization, but probably not relevant
    __cuda_weightedCrossEntropyObjectiveFunction<<<1, 1>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, objFctn, weights);
}

template __global__ void __cuda_weightedCrossEntropyObjectiveFunction<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn, double* weights);
template void            _cuda_weightedCrossEntropyObjectiveFunction<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn, double* weights);
template __global__ void __cuda_weightedCrossEntropyObjectiveFunction<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn, float* weights);
template void            _cuda_weightedCrossEntropyObjectiveFunction<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn, float* weights);

// squaredErrorObjectiveFunction

template<typename T>
__global__ void __cuda_squaredErrorObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn) {
    unsigned int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < nRows) {
        objFctn[row] = 0.0f;
        for (int column = 0; column < nColumns; column++) {
            T            kroneckerDelta = alignmentDevPtr[column] == row ? 1.0 : 0.0;
            unsigned int position       = column * nRows + row;
            objFctn[row] += (matrixPtr[position] - kroneckerDelta) * (matrixPtr[position] - kroneckerDelta);
        }
    }
}

template<typename T>
void _cuda_squaredErrorObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn) {
    unsigned int nElements = nRows;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    // no parallelization, but probably not relevant
    __cuda_squaredErrorObjectiveFunction<<<gridSize, THREADS_PER_BLOCK>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, objFctn);
}

template __global__ void __cuda_squaredErrorObjectiveFunction(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn);
template void            _cuda_squaredErrorObjectiveFunction(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn);
template __global__ void __cuda_squaredErrorObjectiveFunction(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn);
template void            _cuda_squaredErrorObjectiveFunction(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn);

// weightedSquaredErrorObjectiveFunction

template<typename T>
__global__ void __cuda_weightedSquaredErrorObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn, T* weights) {
    unsigned int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < nRows) {
        objFctn[row] = 0.0f;
        for (int column = 0; column < nColumns; column++) {
            T            kroneckerDelta = alignmentDevPtr[column] == row ? 1.0 : 0.0;
            unsigned int position       = column * nRows + row;
            objFctn[row] += (matrixPtr[position] - kroneckerDelta) * (matrixPtr[position] - kroneckerDelta) * weights[column];
        }
    }
}

template<typename T>
void _cuda_weightedSquaredErrorObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn, T* weights) {
    unsigned int nElements = nRows;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_weightedSquaredErrorObjectiveFunction<<<gridSize, THREADS_PER_BLOCK>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, objFctn, weights);
}

template __global__ void __cuda_weightedSquaredErrorObjectiveFunction(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn, double* weights);
template void            _cuda_weightedSquaredErrorObjectiveFunction(double* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, double* objFctn, double* weights);
template __global__ void __cuda_weightedSquaredErrorObjectiveFunction(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn, float* weights);
template void            _cuda_weightedSquaredErrorObjectiveFunction(float* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, float* objFctn, float* weights);

// ###########################################################################
// binaryDivergenceObjectiveFunction
template<typename T>
__global__ void __cuda_binaryDivergenceObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn) {
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns) {
        objFctn[column] = 0.0;
        for (int row = 0; row < nRows; row++) {
            unsigned int position = column * nRows + row;
            if (alignmentDevPtr[column] == row)
                objFctn[column] -= log(matrixPtr[position]);
            else
                objFctn[column] -= log(1.0 - matrixPtr[position]);
        }
    }
}
template<typename T>
void _cuda_binaryDivergenceObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn) {
    int gridSize = (int)ceil((float)nColumns / THREADS_PER_BLOCK);
    __cuda_binaryDivergenceObjectiveFunction<T><<<gridSize, THREADS_PER_BLOCK>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, objFctn);
}
template __global__ void __cuda_binaryDivergenceObjectiveFunction<float>(float*, unsigned int, unsigned int, unsigned int*, float*);
template __global__ void __cuda_binaryDivergenceObjectiveFunction<double>(double*, unsigned int, unsigned int, unsigned int*, double*);
template void            _cuda_binaryDivergenceObjectiveFunction<float>(float*, unsigned int, unsigned int, unsigned int*, float*);
template void            _cuda_binaryDivergenceObjectiveFunction<double>(double*, unsigned int, unsigned int, unsigned int*, double*);

// ###########################################################################
// weightedBinaryDivergenceObjectiveFunction
template<typename T>
__global__ void __cuda_weightedBinaryDivergenceObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn, T* weights) {
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns) {
        objFctn[column] = 0.0;
        for (int row = 0; row < nRows; row++) {
            unsigned int position = column * nRows + row;
            if (alignmentDevPtr[column] == row)
                objFctn[column] -= log(matrixPtr[position]) * weights[column];
            else
                objFctn[column] -= log(1.0 - matrixPtr[position]) * weights[column];
        }
    }
}
template<typename T>
void _cuda_weightedBinaryDivergenceObjectiveFunction(T* matrixPtr, unsigned int nRows, unsigned int nColumns, unsigned int* alignmentDevPtr, T* objFctn, T* weights) {
    int gridSize = (int)ceil((float)nColumns / THREADS_PER_BLOCK);
    __cuda_weightedBinaryDivergenceObjectiveFunction<T><<<gridSize, THREADS_PER_BLOCK>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, objFctn, weights);
}
template __global__ void __cuda_weightedBinaryDivergenceObjectiveFunction<float>(float*, unsigned int, unsigned int, unsigned int*, float*, float*);
template __global__ void __cuda_weightedBinaryDivergenceObjectiveFunction<double>(double*, unsigned int, unsigned int, unsigned int*, double*, double*);
template void            _cuda_weightedBinaryDivergenceObjectiveFunction<float>(float*, unsigned int, unsigned int, unsigned int*, float*, float*);
template void            _cuda_weightedBinaryDivergenceObjectiveFunction<double>(double*, unsigned int, unsigned int, unsigned int*, double*, double*);

// ###########################################################################
// binary divergence softmax gradient computation

template<typename T>
__global__ void __cuda_binaryDivergenceSoftmaxGradient(T* gradient, unsigned int nRows, unsigned int nColumns, const T* output, const unsigned int* alignment) {
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumns) {
        T constsum = 0.0;
        for (int i = 0; i < nRows; ++i) {
            unsigned int position = column * nRows + i;
            const T      y        = output[position];
            if (alignment[column] == i)
                constsum -= 1.0;
            else if (y < 1.0)
                constsum += y / (1.0 - y);
        }

        for (int i = 0; i < nRows; ++i) {
            unsigned int position = column * nRows + i;
            const T      y        = output[position];
            if (alignment[column] == i)
                gradient[position] = -1.0 - y * constsum;
            else {
                if (y < 1.0)
                    gradient[position] = y * (1.0 / (1.0 - y) - constsum);
                else
                    gradient[position] = 0.0;
            }
        }
    }
}
template<typename T>
void _cuda_binaryDivergenceSoftmaxGradient(T* matrixPtr, unsigned int nRows, unsigned int nColumns, const T* outputDevPtr, const unsigned int* alignmentDevPtr) {
    int gridSize = (int)ceil((float)nColumns / THREADS_PER_BLOCK);
    __cuda_binaryDivergenceSoftmaxGradient<T><<<gridSize, THREADS_PER_BLOCK>>>(matrixPtr, nRows, nColumns, outputDevPtr, alignmentDevPtr);
}
template __global__ void __cuda_binaryDivergenceSoftmaxGradient<float>(float*, unsigned int, unsigned int, const float*, const unsigned int*);
template __global__ void __cuda_binaryDivergenceSoftmaxGradient<double>(double*, unsigned int, unsigned int, const double*, const unsigned int*);
template void            _cuda_binaryDivergenceSoftmaxGradient<float>(float*, unsigned int, unsigned int, const float*, const unsigned int*);
template void            _cuda_binaryDivergenceSoftmaxGradient<double>(double*, unsigned int, unsigned int, const double*, const unsigned int*);

template<typename T>
__global__ void __cuda_addKroneckerDelta(T* matrixPtr, unsigned int nRows, unsigned int nColumns, const unsigned int* alignmentDevPtr, const T scale) {
    unsigned int index     = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int nElements = nRows * nColumns;
    if (index < nElements) {
        unsigned int colIndex = index / nRows;
        unsigned int rowIndex = index % nRows;
        matrixPtr[index] += rowIndex == alignmentDevPtr[colIndex] ? scale : 0.0;
    }
}

template<typename T>
void _cuda_addKroneckerDelta(T* matrixPtr, unsigned int nRows, unsigned int nColumns, const unsigned int* alignmentDevPtr, const T scale) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_addKroneckerDelta<<<gridSize, THREADS_PER_BLOCK>>>(matrixPtr, nRows, nColumns, alignmentDevPtr, scale);
}

template __global__ void __cuda_addKroneckerDelta<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, const unsigned int* alignmentDevPtr, const double scale);
template void            _cuda_addKroneckerDelta<double>(double* matrixPtr, unsigned int nRows, unsigned int nColumns, const unsigned int* alignmentDevPtr, const double scale);
template __global__ void __cuda_addKroneckerDelta<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, const unsigned int* alignmentDevPtr, const float scale);
template void            _cuda_addKroneckerDelta<float>(float* matrixPtr, unsigned int nRows, unsigned int nColumns, const unsigned int* alignmentDevPtr, const float scale);

/*
 *  appendSecondOrderFeatures
 */

template<typename T>
__global__ void __cuda_appendSecondOrderFeatures(const T* X, unsigned int nRowsX, unsigned int nColumnsX, T* Y, unsigned int nRowsY, unsigned int offset) {
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumnsX) {
        unsigned int pos = offset;
        for (unsigned int i = 0; i < nRowsX; ++i) {
            for (unsigned int j = i; j < nRowsX; ++j) {
                Y[column * nRowsY + pos] = X[column * nRowsX + i] * X[column * nRowsX + j];
                pos++;
            }
        }
    }
}

template<typename T>
void _cuda_appendSecondOrderFeatures(const T* X, unsigned int nRowsX, unsigned int nColumnsX, T* Y, unsigned int nRowsY, unsigned int offset) {
    int gridSize = (int)ceil((float)nColumnsX / THREADS_PER_BLOCK);
    __cuda_appendSecondOrderFeatures<<<gridSize, THREADS_PER_BLOCK>>>(X, nRowsX, nColumnsX, Y, nRowsY, offset);
}

template __global__ void __cuda_appendSecondOrderFeatures(const double* X, unsigned int nRowsX, unsigned int nColumnsX, double* Y, unsigned int nRowsY, unsigned int offset);
template void            _cuda_appendSecondOrderFeatures(const double* X, unsigned int nRowsX, unsigned int nColumnsX, double* Y, unsigned int nRowsY, unsigned int offset);
template __global__ void __cuda_appendSecondOrderFeatures(const float* X, unsigned int nRowsX, unsigned int nColumnsX, float* Y, unsigned int nRowsY, unsigned int offset);
template void            _cuda_appendSecondOrderFeatures(const float* X, unsigned int nRowsX, unsigned int nColumnsX, float* Y, unsigned int nRowsY, unsigned int offset);

// appendThirdOrderFeatures

template<typename T>
__global__ void __cuda_appendThirdOrderFeatures(const T* X, unsigned int nRowsX, unsigned int nColumnsX, T* Y, unsigned int nRowsY, unsigned int offset) {
    unsigned int column = threadIdx.x + blockIdx.x * blockDim.x;
    if (column < nColumnsX) {
        unsigned int pos = offset;
        for (unsigned int i = 0; i < nRowsX; ++i) {
            for (unsigned int j = i; j < nRowsX; ++j) {
                for (unsigned int k = j; k < nRowsX; ++k) {
                    Y[column * nRowsY + pos] = X[column * nRowsX + i] * X[column * nRowsX + j] * X[column * nRowsX + k];
                    pos++;
                }
            }
        }
    }
}

template<typename T>
void _cuda_appendThirdOrderFeatures(const T* X, unsigned int nRowsX, unsigned int nColumnsX, T* Y, unsigned int nRowsY, unsigned int offset) {
    int gridSize = (int)ceil((float)nColumnsX / THREADS_PER_BLOCK);
    __cuda_appendThirdOrderFeatures<<<gridSize, THREADS_PER_BLOCK>>>(X, nRowsX, nColumnsX, Y, nRowsY, offset);
}

template __global__ void __cuda_appendThirdOrderFeatures(const double* X, unsigned int nRowsX, unsigned int nColumnsX, double* Y, unsigned int nRowsY, unsigned int offset);
template void            _cuda_appendThirdOrderFeatures(const double* X, unsigned int nRowsX, unsigned int nColumnsX, double* Y, unsigned int nRowsY, unsigned int offset);
template __global__ void __cuda_appendThirdOrderFeatures(const float* X, unsigned int nRowsX, unsigned int nColumnsX, float* Y, unsigned int nRowsY, unsigned int offset);
template void            _cuda_appendThirdOrderFeatures(const float* X, unsigned int nRowsX, unsigned int nColumnsX, float* Y, unsigned int nRowsY, unsigned int offset);
/*
 *
 * dropout
 *
 */
template<typename T>
__global__ void __cuda_dropout(T* data, const T* mask, unsigned int nElements, T dropoutProbability) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if ((index < nElements) && (mask[index] <= dropoutProbability))
        data[index] = 0.0;
}

template<typename T>
void _cuda_dropout(T* data, const T* mask, unsigned int nRows, unsigned int nColumns, T dropoutProbability) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_dropout<<<gridSize, THREADS_PER_BLOCK>>>(data, mask, nElements, dropoutProbability);
}

template __global__ void __cuda_dropout(double* data, const double* mask, unsigned int nElements, double dropoutProbability);
template void            _cuda_dropout(double* data, const double* mask, unsigned int nRows, unsigned int nColumns, double dropoutProbability);
template __global__ void __cuda_dropout(float* data, const float* mask, unsigned int nElements, float dropoutProbability);
template void            _cuda_dropout(float* data, const float* mask, unsigned int nRows, unsigned int nColumns, float dropoutProbability);

/*
 *
 * l1clipping
 *
 */
template<typename T>
__global__ void __cuda_l1clipping(T* data, unsigned int nElements, T value) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        if (data[index] > 0) {
            if (data[index] - value > 0)
                data[index] = data[index] - value;
            else
                data[index] = 0;
        }
        else if (data[index] < 0) {
            if (data[index] + value < 0)
                data[index] = data[index] + value;
            else
                data[index] = 0;
        }
    }
}

template<typename T>
void _cuda_l1clipping(T* data, unsigned int nRows, unsigned int nColumns, T value) {
    unsigned int nElements = nRows * nColumns;
    int          gridSize  = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_l1clipping<<<gridSize, THREADS_PER_BLOCK>>>(data, nElements, value);
}

template __global__ void __cuda_l1clipping(double* data, unsigned int nElements, double value);
template void            _cuda_l1clipping(double* data, unsigned int nRows, unsigned int nColumns, double value);
template __global__ void __cuda_l1clipping(float* data, unsigned int nElements, float value);
template void            _cuda_l1clipping(float* data, unsigned int nRows, unsigned int nColumns, float value);

/*
 *
 * clip
 *
 */
template<typename T>
__global__ void __cuda_clip(T* data, unsigned int nElements, T maxAbsValue);

template<>
__global__ void __cuda_clip(float* data, unsigned int nElements, float maxAbsValue) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        data[index] = data[index] > 0 ? fminf(data[index], maxAbsValue) : fmaxf(data[index], -maxAbsValue);
    }
}
template<>
__global__ void __cuda_clip(double* data, unsigned int nElements, double maxAbsValue) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < nElements) {
        data[index] = data[index] > 0 ? fmin(data[index], maxAbsValue) : fmax(data[index], -maxAbsValue);
    }
}

template<typename T>
void _cuda_clip(T* data, unsigned int nElements, T maxAbsValue) {
    int gridSize = (int)ceil((float)nElements / THREADS_PER_BLOCK);

    __cuda_clip<<<gridSize, THREADS_PER_BLOCK>>>(data, nElements, maxAbsValue);
}

// template __global__ void __cuda_clip(double *data, unsigned int nElements, double value);
template void _cuda_clip(double* data, unsigned int nElements, double value);
// template __global__ void __cuda_clip(float *data, unsigned int nElements, float value);
template void _cuda_clip(float* data, unsigned int nElements, float value);

//////////////
// maxout helper functions
// forward

template<typename T>
__global__ void __cuda_addPoolingMax(const T* input, T* output, unsigned int* argmax,
                                     unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out,
                                     unsigned int poolingSize, bool poolingAbs) {
    // unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int pool     = blockIdx.x;
    unsigned int column   = threadIdx.x;
    unsigned int pool_max = nRows_in / poolingSize;
    // if (column < nColumns) {
    if (column < nColumns && pool < pool_max) {
        unsigned int inpos = nRows_in * column + pool * poolingSize;
        // for (unsigned int pool = 0; pool < nRows_in/poolingSize; ++pool) {
        T            maxval = -9999999999;
        T            val;
        unsigned int maxidx = 0;
        for (unsigned int in = 0; in < poolingSize; ++in, ++inpos) {
            // T val = poolingAbs ? abs(input[inpos]) : input[inpos]; // TODO: remove?
            val = input[inpos];
            if (maxval < val) {
                maxval = val;
                maxidx = inpos;
            }
        }
        unsigned int outpos = column * nRows_out + pool;
        output[outpos]      = maxval;
        argmax[outpos]      = maxidx;
        //}
    }
}

template<typename T>
void _cuda_addPoolingMax(const T* input, T* output, unsigned int* argmax,
                         unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out,
                         unsigned int poolingSize, bool poolingAbs) {
    // parallelization over columns only
    // int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    //__cuda_addPoolingMax <<< gridSize , THREADS_PER_BLOCK >>> (input, output, argmax, nColumns, nRows_in, nRows_out, poolingSize, poolingAbs);
    int pool_max = nRows_in / poolingSize;
    __cuda_addPoolingMax<<<pool_max, nColumns>>>(input, output, argmax, nColumns, nRows_in, nRows_out, poolingSize, poolingAbs);
    if (hipSuccess != hipGetLastError())
        printf("Error 8\n");
}

template void _cuda_addPoolingMax(const double* input, double* output, unsigned int* argmax,
                                  unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out, unsigned int poolingSize, bool poolingAbs);
template void _cuda_addPoolingMax(const float* input, float* output, unsigned int* argmax,
                                  unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out, unsigned int poolingSize, bool poolingAbs);

// maxout backward

template<typename T>
__global__ void __cuda_backpropPoolingMax(T* output, const unsigned int* argmax, const T* error,
                                          unsigned int nColumns, unsigned int nRows_err) {
    // unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int err_idx = blockIdx.x;
    unsigned int column  = threadIdx.x;

    // if (column < nColumns) {
    if (column < nColumns && err_idx < nRows_err) {
        unsigned int offset = column * nRows_err;
        // for (unsigned int pos = offset; pos < offset + nRows_err; ++pos) {
        int pos             = offset + err_idx;
        output[argmax[pos]] = error[pos];
        //}
    }
}

template<typename T>
void _cuda_backpropPoolingMax(T* output, const unsigned int* argmax, const T* error,
                              unsigned int nColumns, unsigned int nRows_err) {
    // parallelization over columns only
    // int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    //__cuda_backpropPoolingMax <<< gridSize , THREADS_PER_BLOCK >>> (output, argmax, error, nColumns, nRows_err);
    __cuda_backpropPoolingMax<<<nRows_err, nColumns>>>(output, argmax, error, nColumns, nRows_err);
    if (hipSuccess != hipGetLastError())
        printf("Error 7\n");
}

template void _cuda_backpropPoolingMax(double* output, const unsigned int* argmax, const double* error,
                                       unsigned int nColumns, unsigned int nRows_err);
template void _cuda_backpropPoolingMax(float* output, const unsigned int* argmax, const float* error,
                                       unsigned int nColumns, unsigned int nRows_err);

//////////////
// P-norm pooling
// forward

template<typename T>
__global__ void __cuda_addPoolingPnorm(const T* input, T* output,
                                       unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out,
                                       unsigned int poolingSize, int pnorm) {
    // unsigned  int column = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int pool     = blockIdx.x;
    unsigned int column   = threadIdx.x;
    unsigned int pool_max = nRows_in / poolingSize;
    // if (column < nColumns) {
    if (column < nColumns && pool < pool_max) {
        unsigned int inpos     = nRows_in * column + pool * poolingSize;
        T            inv_pnorm = 1.0 / pnorm;
        // for (unsigned int pool = 0; pool < nRows_in/poolingSize; ++pool) {
        T val = 0;
        for (unsigned int in = 0; in < poolingSize; ++in, ++inpos) {
            // T val = poolingAbs ? abs(input[inpos]) : input[inpos]; // TODO: remove?
            val += pow(abs(input[inpos]), pnorm);
        }
        unsigned int outpos = column * nRows_out + pool;
        output[outpos]      = pow(val, inv_pnorm);
        //}
    }
}

template<typename T>
void _cuda_addPoolingPnorm(const T* input, T* output,
                           unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out,
                           unsigned int poolingSize, unsigned int pnorm) {
    // parallelization over columns only
    // int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    //__cuda_addPoolingMax <<< gridSize , THREADS_PER_BLOCK >>> (input, output, argmax, nColumns, nRows_in, nRows_out, poolingSize, poolingAbs);
    int pool_max = nRows_in / poolingSize;
    __cuda_addPoolingPnorm<<<pool_max, nColumns>>>(input, output, nColumns, nRows_in, nRows_out, poolingSize, pnorm);
    if (hipSuccess != hipGetLastError())
        printf("Error 6\n");
}

template void _cuda_addPoolingPnorm(const double* input, double* output,
                                    unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out, unsigned int poolingSize, unsigned int pnorm);
template void _cuda_addPoolingPnorm(const float* input, float* output,
                                    unsigned int nColumns, unsigned int nRows_in, unsigned int nRows_out, unsigned int poolingSize, unsigned int pnorm);

// maxout backward

template<typename T>
__global__ void __cuda_backpropPoolingPnorm(T* output, const T* error,
                                            unsigned int nColumns, unsigned int nRows_err, unsigned int poolingSize, unsigned int pnorm) {
    unsigned int pool     = blockIdx.x;
    unsigned int column   = threadIdx.x;
    unsigned int nRows_in = nRows_err * poolingSize;
    if (column < nColumns && pool < nRows_err) {
        unsigned int inpos = nRows_in * column + pool * poolingSize;
        for (unsigned int in = 0; in < poolingSize; ++in, ++inpos) {
            int pos       = column * nRows_err + pool;
            output[inpos] = error[pos];
        }
    }
}

template<typename T>
void _cuda_backpropPoolingPnorm(T* output, const T* error,
                                unsigned int nColumns, unsigned int nRows_err,
                                unsigned int poolingSize, unsigned int pnorm) {
    // parallelization over columns only
    // int gridSize = (int)ceil( (float) nColumns/THREADS_PER_BLOCK);
    //__cuda_backpropPoolingMax <<< gridSize , THREADS_PER_BLOCK >>> (output, argmax, error, nColumns, nRows_err);
    __cuda_backpropPoolingPnorm<<<nRows_err, nColumns>>>(output, error, nColumns, nRows_err, poolingSize, pnorm);
    if (hipSuccess != hipGetLastError())
        printf("Error 5\n");
}

template void _cuda_backpropPoolingPnorm(double* output, const double* error,
                                         unsigned int nColumns, unsigned int nRows_err, unsigned int poolingSize, unsigned int pnorm);
template void _cuda_backpropPoolingPnorm(float* output, const float* error,
                                         unsigned int nColumns, unsigned int nRows_err, unsigned int poolingSize, unsigned int pnorm);

// ###########################################################################
// convolutional layer helping functions

template<typename T>
__global__ void __cuda_convExtractPatches(const T* input, const int* patchIdx, T* patches,
                                          int* inverse_patches,
                                          int input_frames, int input_dim, int shifts_num, int shifts_dim, int patch_dim) {
    int t = blockIdx.x;
    // int si = blockIdx.y;
    // int s  = threadIdx.x;
    int s   = blockIdx.y;
    int tid = threadIdx.x;

    // volatile __shared__ int tgt_idx[THREADS_PER_BLOCK];
    // volatile __shared__ T   tgt_val[THREADS_PER_BLOCK];
    // tgt_idx[tid] = -1;
    // tgt_val[tid] = 0;

    int si_block = (int)ceil((float)shifts_dim / THREADS_PER_BLOCK);

    // if (t < input_frames) {
    // if (t < input_frames && s < shifts_num && si < shifts_dim) {
    if (t < input_frames && s < shifts_num) {
        int num_input_elems = input_frames * input_dim;
        int inverse_idx;
        // for (int s = 0; s < shifts_num; ++s) {
        for (int si = tid * si_block; si < (tid + 1) * si_block; ++si) {
            if (si >= shifts_dim)
                break;
            int pos_patches = (t * shifts_num + s) * shifts_dim + si;

            int i = t * input_dim + patchIdx[shifts_num * si + s];  // patchIdx.at(s, si);
            if (i < 0 || i >= num_input_elems)
                continue;
            // while (i < 0)               i += input_dim;
            // while (i >= num_input_elems) i -= input_dim;

            T val = 0;
            // if (i >= 0 && i < num_input_elems)  {

            val = input[i];
            // inverse_idx = inverse_patches_counts[i]*num_input_elems + i;
            inverse_idx                  = (si % patch_dim) * num_input_elems + i;
            inverse_patches[inverse_idx] = pos_patches;  // slow TODO
                                                         // if (pos_patches < 0 || pos_patches >= 138240)
            // printf("%d: %d -> %d, %d, %d\n", inverse_patches_counts[i], inverse_idx, pos_patches, i, t);
            // inverse_patches_counts[i] += 1;
            // atomicAdd(&(inverse_patches_counts[i]), 1);
            //}
            // patches->at(si, t*shifts_num + s) = val;

            patches[pos_patches] = val;  // slow TODO
                                         // tgt_idx[tid] = pos_patches;
            // tgt_val[tid] = val;
            // if(t==1 && s == 0) printf("t=%d s=%d si=%d i=%d patchIdx=%d pos_patches=%d, inverse_idx=%d\n", t, s, si, i, patchIdx[shifts_num*si + s], pos_patches, inverse_idx);
        }
        //}
        //__syncthreads();
        // int FACTOR = patch_dim;
        // if (tid < FACTOR) {
        //    for (int i = 0; i < THREADS_PER_BLOCK; i += FACTOR) {
        //        if(tgt_idx[i] > -1)
        //            patches[tgt_idx[i]] = tgt_val[i];
        //    }
        //}
    }
}
template<typename T>
void _cuda_convExtractPatches(const T* input, const int* patchIdx, T* patches,
                              int* inverse_patches,
                              int input_frames, int input_dim, int shifts_num, int shifts_dim, int patch_dim) {
    //    dim3 gridSize(input_frames, shifts_dim);
    //    __cuda_convExtractPatches<T> <<<gridSize, shifts_num>>> (input, patchIdx, patches,
    //	        inverse_patches,
    //		input_frames, input_dim, shifts_num, shifts_dim, patch_dim);
    dim3 gridSize(input_frames, shifts_num);
    __cuda_convExtractPatches<T><<<gridSize, THREADS_PER_BLOCK>>>(input, patchIdx, patches,
                                                                  inverse_patches,
                                                                  input_frames, input_dim, shifts_num, shifts_dim, patch_dim);
    if (hipSuccess != hipGetLastError())
        printf("Error 4\n");
}

template void _cuda_convExtractPatches(const float* input, const int* patchIdx, float* patches,
                                       int* inverse_patches,
                                       int input_frames, int input_dim, int shifts_num, int shifts_dim, int patch_dim);
template void _cuda_convExtractPatches(const double* input, const int* patchIdx, double* patches,
                                       int* inverse_patches,
                                       int input_frames, int input_dim, int shifts_num, int shifts_dim, int patch_dim);

///////////////////////////////////////

template<typename T>
__global__ void __cuda_convRestoreFromPatches(T* unwarped_error, const T* warped_error,
                                              const int* patchIdx, int* inverse_patches, int num_input_elems, int patch_dim) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < num_input_elems) {
        //	T err = 0;
        int idx;
        int inv_patch_idx;
        /*
        if (i==0) {
            for(int a=0; a<1000; ++a) {
                for(int b=0; b<42; ++b) {
                    printf("%d ", inverse_patches[b*num_input_elems+a] );
                }
                printf("\n");
            }
        }
        */
        // int N = 0;
        for (int col = 0; col < patch_dim; ++col) {
            inv_patch_idx = col * num_input_elems + i;
            idx           = inverse_patches[inv_patch_idx];
            // if (i==0) printf("col=%d, inv_patch_idx=%d, idx=%d \n", col, inv_patch_idx, idx);
            /*
                        if (col>=81 || col*num_input_elems+i >= 1244160 || idx >= 131328 || idx < 0 || i >= 15360) {
                            //for (int c=0; c<inverse_patches_counts[i]; ++c)
                                printf("c=%d idx=%d i=%d \n", col, inverse_patches[col*num_input_elems+i], i);
                            //printf("\n");

                        //printf("i=%d col=%d/%d, idx=%d, err=%f\n", i, col, inverse_patches_counts[i], idx, err);
                        }
                        */

            if (idx > 0) {
                unwarped_error[i] += warped_error[idx];
                inverse_patches[inv_patch_idx] = 0;
                //++N;
            }
        }
        // if (inverse_patches_counts[i]>0) unwarped_error[i] /= inverse_patches_counts[i];
        // else printf("WTF?!");
        // unwarped_error[i] /= N;
    }
}
template<typename T>
void _cuda_convRestoreFromPatches(T* unwarped_error, const T* warped_error,
                                  const int* patchIdx, int* inverse_patches, int num_input_elems, int patch_dim) {
    int gridSize = (int)ceil((float)num_input_elems / THREADS_PER_BLOCK);
    __cuda_convRestoreFromPatches<T><<<gridSize, THREADS_PER_BLOCK>>>(unwarped_error, warped_error, patchIdx,
                                                                      inverse_patches, num_input_elems, patch_dim);
    if (hipSuccess != hipGetLastError())
        printf("Error 3\n");
}

template void _cuda_convRestoreFromPatches(float* unwarped_error, const float* warped_error,
                                           const int* patchIdx, int* inverse_patches,
                                           int num_input_elems, int patch_dim);
template void _cuda_convRestoreFromPatches(double* unwarped_error, const double* warped_error,
                                           const int* patchIdx, int* inverse_patches,
                                           int num_input_elems, int patch_dim);

///////////////////////////////////////
template<typename T>
__global__ void __cuda_convUnwarpFrames(const T* output_warped, const T* bias, T* output,
                                        int output_dim, int filter_num, int shifts_num, int input_frames) {
    // int t = threadIdx.x + blockIdx.x * blockDim.x;
    int t = blockIdx.x;
    int f = threadIdx.x;
    if (t < input_frames && f < filter_num) {
        // int num_input_elems = input_frames * input_dim;
        // int row = 0;
        int row = f * shifts_num;
        // for (int f = 0; f < filter_num; ++f) {
        for (int s = 0; s < shifts_num; ++s) {
            int i = (t * shifts_num + s) * filter_num + f;

            output[t * output_dim + row] = output_warped[i] + bias[f];
            ++row;
        }
        //}
    }
}
template<typename T>
void _cuda_convUnwarpFrames(const T* output_warped, const T* bias, T* output,
                            int output_dim, int filter_num, int shifts_num, int input_frames) {
    // int gridSize = (int)ceil( (float) input_frames/THREADS_PER_BLOCK);
    //__cuda_convUnwarpFrames<T> <<<gridSize, THREADS_PER_BLOCK>>> (output_warped, bias, output, output_dim, filter_num, shifts_num, input_frames);
    __cuda_convUnwarpFrames<T><<<input_frames, filter_num>>>(output_warped, bias, output, output_dim, filter_num, shifts_num, input_frames);
    if (hipSuccess != hipGetLastError())
        printf("Error 2\n");
}

template void _cuda_convUnwarpFrames(const float* output_warped, const float* bias, float* output,
                                     int output_dim, int filter_num, int shifts_num, int input_frames);
template void _cuda_convUnwarpFrames(const double* output_warped, const double* bias, double* output,
                                     int output_dim, int filter_num, int shifts_num, int input_frames);

///////////////////////////////////////

template<typename T>
__global__ void __cuda_convWarpFrames(const T* error_unwarped, T* output,
                                      int error_dim, int filter_num, int shifts_num, int input_frames) {
    // int t = threadIdx.x + blockIdx.x * blockDim.x;
    // if (t < input_frames) {
    int t = blockIdx.x;
    int f = threadIdx.x;

    if (t < input_frames && f < filter_num) {
        // int num_input_elems = input_frames * input_dim;
        int row = f * shifts_num;
        // for (int f = 0; f < filter_num; ++f) {
        for (int s = 0; s < shifts_num; ++s) {
            int i = (t * shifts_num + s) * filter_num + f;

            output[i] = error_unwarped[t * error_dim + row];
            ++row;
            /*
            if (t==0 && s==0) {
                printf("%d %d %d -> %f\n", f, i, row, output[i]);
            }
            */
        }
        //}
    }
}
template<typename T>
void _cuda_convWarpFrames(const T* error_unwarped, T* output,
                          int error_dim, int filter_num, int shifts_num, int input_frames) {
    // int gridSize = (int)ceil( (float) input_frames/THREADS_PER_BLOCK);
    //__cuda_convWarpFrames<T> <<<gridSize, THREADS_PER_BLOCK>>> (error_unwarped, output, error_dim, filter_num, shifts_num, input_frames);
    __cuda_convWarpFrames<T><<<input_frames, filter_num>>>(error_unwarped, output, error_dim, filter_num, shifts_num, input_frames);
    if (hipSuccess != hipGetLastError())
        printf("Error 1\n");
}

template void _cuda_convWarpFrames(const float* error_unwarped, float* output,
                                   int error_dim, int filter_num, int shifts_num, int input_frames);
template void _cuda_convWarpFrames(const double* error_unwarped, double* output,
                                   int error_dim, int filter_num, int shifts_num, int input_frames);
